#include "cgls.h"
#include "cgls.cuh"

unsigned cudacgls(const value *val, const unsigned *ptr, const unsigned *ind, const unsigned m,
		  const unsigned n, const unsigned nnz, const value *b, value *x) {

	value *d_val, *d_b, *d_x;
	int *d_ptr, *d_ind;

	hipMallocManaged(&d_val, sizeof(value) * nnz);
	hipMallocManaged(&d_ptr, sizeof(int) * n + 1);
	hipMallocManaged(&d_ind, sizeof(int) * nnz);
	hipMallocManaged(&d_b, sizeof(value) * m);
	hipMallocManaged(&d_x, sizeof(value) * n);

	unsigned rc = cgls::Solve<value, cgls::CSC>(d_val, d_ptr, d_ind, (int)m, (int)n, (int)nnz,
						    d_b, d_x, 0, TOLERANCE, MAXITERATIONS, !CGLSDEBUG);

	hipFree(d_val);
	hipFree(d_ptr);
	hipFree(d_ind);
	hipFree(d_b);
	hipFree(d_x);

	return rc;
}
