#include "cgls.h"
#include "cgls.cuh"

unsigned cudacgls(const value *val, const unsigned *ptr, const unsigned *ind, const unsigned m,
		  const unsigned n, const unsigned nnz, value *b, value *x) {

	value *d_val, *d_b, *d_x;
	int *d_ptr, *d_ind;

	hipMallocManaged(&d_val, sizeof(value) * nnz);
	hipMallocManaged(&d_ptr, sizeof(int) * n + 1);
	hipMallocManaged(&d_ind, sizeof(int) * nnz);
	hipMallocManaged(&d_b, sizeof(value) * m);
	hipMallocManaged(&d_x, sizeof(value) * n);

	memcpy(d_val, val, sizeof(value) * nnz);
	memcpy(d_ptr, ptr, sizeof(int) * n + 1);
	memcpy(d_ind, ind, sizeof(int) * nnz);
	memcpy(d_b, b, sizeof(value) * m);
	memset(d_x, 0, sizeof(value) * n);

	unsigned rc = cgls::Solve<value, cgls::CSC>(d_val, d_ptr, d_ind, m, n, nnz, d_b, d_x,
						    0, TOLERANCE, MAXITERATIONS, !CGLSDEBUG);

	// Store vector of differences in b
	// b = A * x - b

	cgls::Spmv<value, cgls::CSC> spA(m, n, nnz, d_val, d_ptr, d_ind);
	spA('n', 1, d_x, -1, d_b);

	hipDeviceSynchronize();

	memcpy(b, d_b, sizeof(value) * m);
	memcpy(x, d_x, sizeof(value) * n);

	hipFree(d_val);
	hipFree(d_ptr);
	hipFree(d_ind);
	hipFree(d_b);
	hipFree(d_x);

	if (CGLSDEBUG) puts("");

	return rc;
}
