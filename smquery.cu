// CUDA Compute Capabilities Query


#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("sm_%d%d\n", devProp.major, devProp.minor);
	return 0;
}
