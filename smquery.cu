// CUDA Compute Capabilities Query


#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

	// Number of CUDA devices
	int devCount;
	hipGetDeviceCount(&devCount);

	// Iterate through devices
	for (int i = 0; i < devCount; ++i) {
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printf("sm_%d%d\n", devProp.major, devProp.minor);
	}

	return 0;
}
